#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stack>
#include <algorithm>

#include "kernels.cuh"

#include "hip/hip_runtime.h"
#include ""

const int WIDTH = 800;
const int HEIGHT = 600;

void checkCUDAError(const char *);
void generatePointCloud(point *pts, int size, float bottomLX, float bottomLY, float squareSize);

float crossZ(point p1, point p2, point p3)
{
    return (p2.x - p1.x) * (p3.y - p1.y) - (p2.y - p1.y) * (p3.x - p1.x);
}

std::stack<point> grahamScanCPU(point *pts)
{
    int i;
    int min_pt_index = 0;

    // Find index of minimum point
    for (i = 1; i < NUM_POINTS; ++i)
    {
        if (pts[i].y < pts[min_pt_index].y)
        {
            min_pt_index = i;
        }
        else if (pts[i].y == pts[min_pt_index].y) // TODO: better equality test for floats
        {
            if (pts[i].x < pts[min_pt_index].x || pts[i].x == pts[min_pt_index].x) // TODO: better equality test for floats
            {
                min_pt_index = i;
            }
        }
    }

    // assign minimum point p0
    point p0;
    p0.x = pts[min_pt_index].x;
    p0.y = pts[min_pt_index].y;

    // calculate the angle associated with each points vector from p0
    point unit_x;
    unit_x.x = 1;
    unit_x.y = 0;

    point pt;
    point v;
    float len_v;
    float cos_theta;
    for (i = 1; i < NUM_POINTS; ++i)
    {
        if (i != min_pt_index)
        {
            pt.x = pts[i].x;
            pt.y = pts[i].y;
            v.x = pt.x - p0.x;
            v.y = pt.y = p0.y;
            len_v = pow((pow(v.x, 2) + pow(v.y, 2)), 0.5);
            cos_theta = (v.x * unit_x.x + v.y * unit_x.y) / len_v;
            pts[i].angle = cos_theta;
        }
    }

    for (i = 0; i < NUM_POINTS; ++i)
    {
        bool min_pt_found = false;
        if (i == min_pt_index)
        {
            min_pt_found = true;
        }
        else if (min_pt_found)
        {
            pts[i - 1].x = pts[i].x;
            pts[i - 1].y = pts[i].y;
        }
    }

    // sort points by cos angle (using built in to start, maybe should make our own CPU sort?)
    std::sort(pts, pts + NUM_POINTS - 1); // ignoring last point, is no longer relevant after shift

    std::stack<point> s;
    s.push(p0);
    s.push(pts[0]);
    s.push(pts[1]);
    for (int j = 2; j < NUM_POINTS - 1; ++j)
    {
        point pj = pts[j];
        point top = s.top();
        s.pop();
        point next_top = s.top();
        s.pop();
        s.push(next_top);
        s.push(top);
        float cross_z = crossZ(pj, top, next_top);
        while (cross_z < 0)
        {
            s.pop();
            point top = s.top();
            s.pop();
            point next_top = s.top();
            s.pop();
            s.push(next_top);
            s.push(top);
            cross_z = crossZ(pj, top, next_top);
        }
        s.push(pj);
    }
    return s;
}

// function for generating random point cloud
// Generates an array of type point
void generatePointCloud(point *pts, int size, float bottomLX, float bottomLY, float squareSize){
    for (int i = 0; i<size; ++i){
        pts[i].x = bottomLX + static_cast<float>(rand())/RAND_MAX * squareSize;
        pts[i].y = bottomLY + static_cast<float>(rand())/RAND_MAX * squareSize;
    }
}

// TODO: function for rendering point cloud with convex hull

int main(void)
{
    // TODO
    point pointsArray[NUM_POINTS];

    int size = NUM_POINTS;    // Size of the point cloud
    float bottomLeftX = 0.0f; // Bottom left corner of the square
    float bottomLeftY = 0.0f;
    float squareSize = 10.0f; // Size of the square containing the point cloud

    generatePointCloud(pointsArray, size, bottomLeftX, bottomLeftY, squareSize);

    std::stack<point> s = grahamScanCPU(pointsArray);
    point pt;
    while (!s.empty())
    {
        pt = s.top();
        s.pop();
        printf("stack point (%f, %f)", pt.x, pt.y);
    }
}