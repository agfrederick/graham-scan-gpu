#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stack>
#include <algorithm>
#include <cstdlib> // For rand()
#include <GL/glut.h>

#include "kernels.cuh"

#include "hip/hip_runtime.h"
#include ""

const int WIDTH = 800;
const int HEIGHT = 600;

int SIZE = NUM_POINTS;    // Size of the point cloud
float BOTTOMLEFTX = 0.0f; // Bottom left corner of the square
float BOTTOMLEFTY = 0.0f;
float SQUARESIZE = 10.0f; // Size of the square containing the point cloud

void checkCUDAError(const char *);
void generatePointCloud(point *pts, int size, float bottomLX, float bottomLY, float squareSize);
point minPointGPU(points *h_points, points *h_points_result, points *d_points, points *d_points_result);
void calculateCosAnglesGPU(points *h_points, points *d_points, point p0);

float crossZ(point p1, point p2, point p3)
{
    return (p2.x - p1.x) * (p3.y - p1.y) - (p2.y - p1.y) * (p3.x - p1.x);
}

void pointArrayToPoints(point *pts, points *output)
{
    for (int i = 0; i < NUM_POINTS; ++i)
    {
        // get values from array of structs
        float x = pts[i].x;
        float y = pts[i].y;

        // assign values to struct arrays
        output->x[i] = x;
        output->y[i] = y;
    }
}

std::stack<point> grahamScanCPU(point *pts)
{
    int i;
    int min_pt_index = 0;

    // Find index of minimum point
    for (i = 1; i < NUM_POINTS; ++i)
    {
        if (pts[i].y < pts[min_pt_index].y)
        {
            min_pt_index = i;
        }
        else if (pts[i].y == pts[min_pt_index].y) // TODO: better equality test for floats
        {
            if (pts[i].x < pts[min_pt_index].x || pts[i].x == pts[min_pt_index].x) // TODO: better equality test for floats
            {
                min_pt_index = i;
            }
        }
    }

    // assign minimum point p0
    point p0;
    p0.x = pts[min_pt_index].x;
    p0.y = pts[min_pt_index].y;

    // calculate the angle associated with each points vector from p0
    point unit_x;
    unit_x.x = 1;
    unit_x.y = 0;

    point pt;
    point v;
    float len_v;
    float cos_theta;
    for (i = 1; i < NUM_POINTS; ++i)
    {
        if (i != min_pt_index)
        {
            pt.x = pts[i].x;
            pt.y = pts[i].y;
            v.x = pt.x - p0.x;
            v.y = pt.y = p0.y;
            len_v = pow((pow(v.x, 2) + pow(v.y, 2)), 0.5);
            cos_theta = (v.x * unit_x.x + v.y * unit_x.y) / len_v;
            pts[i].angle = cos_theta;
        }
    }

    for (i = 0; i < NUM_POINTS; ++i)
    {
        bool min_pt_found = false;
        if (i == min_pt_index)
        {
            min_pt_found = true;
        }
        else if (min_pt_found)
        {
            pts[i - 1].x = pts[i].x;
            pts[i - 1].y = pts[i].y;
        }
    }

    // sort points by cos angle (using built in to start, maybe should make our own CPU sort?)
    std::sort(pts, pts + NUM_POINTS - 1); // ignoring last point, is no longer relevant after shift

    std::stack<point> s;
    s.push(p0);
    s.push(pts[0]);
    s.push(pts[1]);
    for (int j = 2; j < NUM_POINTS - 1; ++j)
    {
        point pj = pts[j];
        point top = s.top();
        s.pop();
        point next_top = s.top();
        s.pop();
        s.push(next_top);
        s.push(top);
        float cross_z = crossZ(pj, top, next_top);
        while (cross_z < 0)
        {
            s.pop();
            point top = s.top();
            s.pop();
            point next_top = s.top();
            s.pop();
            s.push(next_top);
            s.push(top);
            cross_z = crossZ(pj, top, next_top);
        }
        s.push(pj);
    }
    return s;
}

// function for generating random point cloud
// Generates an array of type point
void generatePointCloud(point *pts, int size, float bottomLX, float bottomLY, float squareSize)
{
    for (int i = 0; i < size; ++i)
    {
        pts[i].x = bottomLX + static_cast<float>(rand()) / RAND_MAX * squareSize;
        pts[i].y = bottomLY + static_cast<float>(rand()) / RAND_MAX * squareSize;
    }
}

// TODO: function for rendering point cloud with convex hull
// void renderConvexHull(point *pts, std::stack<point> s)
// {
//     glClear(GL_COLOR_BUFFER_BIT); // Clear the color buffer

//     // Set color
//     glColor3f(1.0f, 0.0f, 0.0f);

//     glBegin(GL_POINTS);

//     // Iterate through the array of points and draw each point
//     for (int i = 0; i < NUM_POINTS; ++i)
//     {
//         glVertex2f(pts[i].x, pts[i].y);
//     }

//     glEnd();

//     glFlush(); // Flush OpenGL pipeline

//     glColor3f(1.0, 0.0, 0.0); // Red color

//     // Begin drawing lines
//     glBegin(GL_LINES);

//     point pt;
//     while (!s.empty())
//     {
//         pt = s.top();
//         s.pop();
//         glVertex2f(pt.x, pt.y);
//     }

//     glEnd();

//     // Flush OpenGL buffer to display the line
//     glFlush();

// Save to file
// unsigned char *pixels = new unsigned char[3 * WIDTH * HEIGHT];
// glReadPixels(0, 0, WIDTH, HEIGHT, GL_RGB, GL_UNSIGNED_BYTE, pixels);

// std::ofstream out("plot.ppm", std::ios::binary);
// out << "P6\n"
//     << WIDTH << " " << HEIGHT << "\n255\n";
// out.write(reinterpret_cast<char *>(pixels), 3 * WIDTH * HEIGHT);
// out.close();

// delete[] pixels;
// }

std::stack<point> grahamScanGPU(point *pts)
{
    points *h_points;
    points *h_points_result;
    points *d_points;
    points *d_points_result;

    h_points = (points *)malloc(sizeof(points));
    h_points_result = (points *)malloc(sizeof(points));
    hipMalloc((void **)&d_points, sizeof(points));
    hipMalloc((void **)&d_points_result, sizeof(points));

    // Generate points
    point pointsArray[NUM_POINTS];

    generatePointCloud(pointsArray, SIZE, BOTTOMLEFTX, BOTTOMLEFTY, SQUARESIZE);

    pointArrayToPoints(pointsArray, h_points);

    // Find minimum point
    point p0 = minPointGPU(h_points, h_points_result, d_points, d_points_result);

    // calculate cos angle with p0 for each point
    calculateCosAnglesGPU(h_points, d_points, p0);

    // sort points using cosine angle and min point

    std::stack<point> s;

    return s;
}

point minPointGPU(points *h_points, points *h_points_result, points *d_points, points *d_points_result)
{
    unsigned int i;
    point min_pt;
    float time;
    hipEvent_t start, stop;

    float max;
    if (BOTTOMLEFTX < BOTTOMLEFTY)
    {
        max = BOTTOMLEFTY + SIZE;
    }
    else
    {
        max = BOTTOMLEFTX + SIZE;
    }

    min_pt.x = max;
    min_pt.y = max;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // memory copy records to device
    hipMemcpy(d_points, h_points, sizeof(points), hipMemcpyHostToDevice);
    checkCUDAError("Min point: CUDA memcpy");

    hipEventRecord(start, 0);
    int blocks = NUM_POINTS / THREADS_PER_BLOCK;
    dim3 numBlocks(blocks);
    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    lowestPoint_kernel<<<numBlocks, threadsPerBlock>>>(d_points, d_points_result);
    hipDeviceSynchronize();
    hipMemcpy(h_points_result, d_points_result, sizeof(points), hipMemcpyDeviceToHost);

    // Reduce the block level results on CPU
    for (int i = 0; i < blocks; ++i)
    {
        float x = h_points_result->x[i];
        float y = h_points_result->y[i];
        if (y < min_pt.y)
        {
            min_pt.x = x;
            min_pt.y = y;
        }
        else if (y == min_pt.y) // TODO: float comparison
        {
            if (x < min_pt.x)
            {
                min_pt.x = x;
                min_pt.y = y;
            }
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    // output result
    printf("GPU lowest point was found at %f, %f\n", min_pt.x, min_pt.y);
    printf("\tExecution time was %f ms\n", time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return min_pt;
}

void calculateCosAnglesGPU(points *h_points, points *d_points, point p0)
{
    unsigned int i;
    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // memory copy records to device
    hipMemcpy(d_points, h_points, sizeof(points), hipMemcpyHostToDevice);
    checkCUDAError("anlges: CUDA memcpy");

    hipEventRecord(start, 0);
    int blocks = NUM_POINTS / THREADS_PER_BLOCK;
    dim3 numBlocks(blocks);
    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    findCosAngles_kernel<<<numBlocks, threadsPerBlock>>>(d_points, p0);
    hipDeviceSynchronize();
    hipMemcpy(h_points, d_points, sizeof(points), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    // output result
    printf("\tExecution time for angle finding was %f ms\n", time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv)
{
    // glutInit(&argc, argv);
    // glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
    // glutInitWindowSize(WIDTH, HEIGHT);
    // glutCreateWindow("OpenGL Plot");
    // glutDisplayFunc(renderConvexHull);
    // glClearColor(0.0, 0.0, 0.0, 1.0);

    // // // Set up the projection matrix
    // // glMatrixMode(GL_PROJECTION);
    // // glLoadIdentity();
    // // gluOrtho2D(-1.0, 1.0, -1.0, 1.0);

    // // Start the GLUT main loop
    // glutMainLoop();
    // TODO
    point pointsArray[NUM_POINTS];

    generatePointCloud(pointsArray, SIZE, BOTTOMLEFTX, BOTTOMLEFTY, SQUARESIZE);

    std::stack<point> s_cpu = grahamScanCPU(pointsArray);
    point pt;
    while (!s_cpu.empty())
    {
        pt = s_cpu.top();
        s_cpu.pop();
        printf("CPU stack point (%f, %f)\n", pt.x, pt.y);
    }

    std::stack<point> s_gpu = grahamScanGPU(pointsArray);
    while (!s_gpu.empty())
    {
        pt = s_gpu.top();
        s_gpu.pop();
        printf("GPU stack point (%f, %f)\n", pt.x, pt.y);
    }
}